/*********************************************************************
 * Filename:   des_test.c
 * Author:     Brad Conte (brad AT bradconte.com)
 * Copyright:
 * Disclaimer: This code is presented "as is" without any guarantees.
 * Details:    Performs known-answer tests on the corresponding DES
 implementation. These tests do not encompass the full
 range of available test vectors, however, if the tests
 pass it is very, very likely that the code is correct
 and was compiled properly. This code also serves as
 example usage of the functions.
 *********************************************************************/

/*************************** HEADER FILES ***************************/
#include <stdio.h>
#include <stdlib.h>
#include <memory.h>
#include <sys/stat.h>
#include <fcntl.h>
//#include <unistd.h>
#include "des.h"
#include <hip/hip_runtime.h>

/*********************** FUNCTION DEFINITIONS ***********************/
int des_test()
{
	BYTE pt1[DES_BLOCK_SIZE] = {0x01,0x23,0x45,0x67,0x89,0xAB,0xCD,0xE7};
	BYTE pt2[DES_BLOCK_SIZE] = {0x01,0x23,0x45,0x67,0x89,0xAB,0xCD,0xEF};
	BYTE ct1[DES_BLOCK_SIZE] = {0xc9,0x57,0x44,0x25,0x6a,0x5e,0xd3,0x1d};
	BYTE ct2[DES_BLOCK_SIZE] = {0x85,0xe8,0x13,0x54,0x0f,0x0a,0xb4,0x05};
	BYTE key1[DES_BLOCK_SIZE] = {0x01,0x23,0x45,0x67,0x89,0xAB,0xCD,0xEF};
	BYTE key2[DES_BLOCK_SIZE] = {0x13,0x34,0x57,0x79,0x9B,0xBC,0xDF,0xF1};

	BYTE schedule[16][6];
	BYTE buf[DES_BLOCK_SIZE];

	BYTE *d_pt1;
	hipMalloc((void **)&d_pt1, DES_BLOCK_SIZE);
	hipMemcpy(d_pt1, pt1, DES_BLOCK_SIZE, hipMemcpyHostToDevice);

	BYTE *d_ct1;
	hipMalloc((void **)&d_ct1, DES_BLOCK_SIZE);
	hipMemcpy(d_ct1, ct1, DES_BLOCK_SIZE, hipMemcpyHostToDevice);

	BYTE *d_pt2;
	hipMalloc((void **)&d_pt2, DES_BLOCK_SIZE);
	hipMemcpy(d_pt2, pt2, DES_BLOCK_SIZE, hipMemcpyHostToDevice);

	BYTE *d_ct2;
	hipMalloc((void **)&d_ct2, DES_BLOCK_SIZE);
	hipMemcpy(d_ct2, ct2, DES_BLOCK_SIZE, hipMemcpyHostToDevice);

	BYTE *d_schedule;
	hipMalloc((void **)&d_schedule, sizeof(schedule));

	BYTE *d_buf;
	hipMalloc((void **)&d_buf, DES_BLOCK_SIZE);

	int pass = 1;

	des_key_setup(key1, schedule, DES_ENCRYPT);
	hipMemcpy(d_schedule, schedule, sizeof(schedule), hipMemcpyHostToDevice);
	des_crypt<<<1, 1>>>(d_pt1, d_buf, d_schedule, DES_BLOCK_SIZE);
	hipMemcpy(buf, d_buf, DES_BLOCK_SIZE, hipMemcpyDeviceToHost);
	pass = pass && !memcmp(ct1, buf, DES_BLOCK_SIZE);

	des_key_setup(key1, schedule, DES_DECRYPT);
	hipMemcpy(d_schedule, schedule, sizeof(schedule), hipMemcpyHostToDevice);
	des_crypt<<<1, 1>>>(d_ct1, d_buf, d_schedule, DES_BLOCK_SIZE);
	hipMemcpy(buf, d_buf, DES_BLOCK_SIZE, hipMemcpyDeviceToHost);
	pass = pass && !memcmp(pt1, buf, DES_BLOCK_SIZE);

	des_key_setup(key2, schedule, DES_ENCRYPT);
	hipMemcpy(d_schedule, schedule, sizeof(schedule), hipMemcpyHostToDevice);
	des_crypt<<<1, 1>>>(d_pt2, d_buf, d_schedule, DES_BLOCK_SIZE);
	hipMemcpy(buf, d_buf, DES_BLOCK_SIZE, hipMemcpyDeviceToHost);
	pass = pass && !memcmp(ct2, buf, DES_BLOCK_SIZE);

	des_key_setup(key2, schedule, DES_DECRYPT);
	hipMemcpy(d_schedule, schedule, sizeof(schedule), hipMemcpyHostToDevice);
	des_crypt<<<1, 1>>>(d_ct2, d_buf, d_schedule, DES_BLOCK_SIZE);
	hipMemcpy(buf, d_buf, DES_BLOCK_SIZE, hipMemcpyDeviceToHost);
	pass = pass && !memcmp(pt2, buf, DES_BLOCK_SIZE);

	return(pass);
}

void enc_dec_file(char *filename)
{
	BYTE *data; // ponteiro para os dados da imagem base que será criptografada.
	BYTE *encrypted_data; // ponteiro para os dados criptografados
	BYTE *decrypted_data; // ponteiro para os dados descriptografados
	//char *filename = "../../sample_files/king_james_bible.txt"; // endereço do arquivo que será utilizado como base

	BYTE key1[DES_BLOCK_SIZE] = {0x01,0x23,0x45,0x67,0x89,0xAB,0xCD,0xEF}; // chave de 64 bits
	BYTE schedule[16][6];

	struct stat st; // stat guarda informações sobre arquivos
	size_t data_size_bytes = 0;

	if (stat(filename, &st) == 0){ // provavelmente verifica se é possivel atribuir o stat st ao arquivo base, e caso sim
		data_size_bytes = sizeof(BYTE) * st.st_size;
		data = (BYTE *) malloc(data_size_bytes); // reserva o tamanho do arquivo base em memória para o ponteiro data.
	};

	FILE *file = fopen(filename, "rb"); // abre o arquivo do tipo nao texto ("rb")

	// copia a imagem para o vetor data
	if(data != NULL && file){ // se o ponteiro data e de arquivo ja foram alocados e abertos, respectivamente, entao prossegue
		int current_byte = 0; // define o byte atual

		// de byte em byte, copia o arquivo para o array data
		while(fread(&data[current_byte], sizeof(BYTE), 1, file) == 1){ 
			current_byte += 1; // atualiza o byte atual
		};
	};

	encrypted_data = (BYTE *) malloc(data_size_bytes); // reserva espaço em memoria para o arquivo criptogrfado
	decrypted_data = (BYTE *) malloc(data_size_bytes); // o mesmo para o arquivo decodificado

	BYTE *d_data; // ponteiro para os dados da imagem base no device
	BYTE *d_encrypted_data; // ponteiro para os dados criptografados no device
	BYTE *d_decrypted_data; // ponteiro para os dados descriptografados no device

	// alocação de memória para os devices
	hipMalloc((void **)&d_data, data_size_bytes);
	hipMalloc((void **)&d_encrypted_data, data_size_bytes);
	hipMalloc((void **)&d_decrypted_data, data_size_bytes);

	hipMemcpy(d_data, data, data_size_bytes, hipMemcpyHostToDevice); // copia o array de dados para o device

	BYTE *d_schedule;
	hipMalloc((void **)&d_schedule, sizeof(schedule));

	int threadsPerBlock = 256;
	int blocksPerGrid =((data_size_bytes + threadsPerBlock - 1) / threadsPerBlock)/DES_BLOCK_SIZE;

	//des_key_setup<<<blocksPerGrid, threadsPerBlock>>>(key1, schedule, DES_ENCRYPT); // seta o schedule para encriptação
	des_key_setup(key1, schedule, DES_ENCRYPT); // seta o schedule para encriptação    
	hipMemcpy(d_schedule, schedule, sizeof(schedule), hipMemcpyHostToDevice);
	des_crypt<<<blocksPerGrid, threadsPerBlock>>>(d_data, d_encrypted_data, d_schedule, data_size_bytes); // criptografa o buffer e salva no data_enc

	//des_key_setup<<<blocksPerGrid, threadsPerBlock>>>(key1, schedule, DES_DECRYPT); // seta o schedule para desencriptação
	des_key_setup(key1, schedule, DES_DECRYPT); // seta o schedule para desencriptação
	hipMemcpy(d_schedule, schedule, sizeof(schedule), hipMemcpyHostToDevice);
	des_crypt<<<blocksPerGrid, threadsPerBlock>>>(d_encrypted_data, d_decrypted_data, d_schedule, data_size_bytes); // descriptografa o data_enc e salva no data_dec

	hipMemcpy(encrypted_data, d_encrypted_data, data_size_bytes, hipMemcpyDeviceToHost); 
	hipMemcpy(decrypted_data, d_decrypted_data, data_size_bytes, hipMemcpyDeviceToHost);

	// cria arquivos para a escrita dos dados cripto e descripto
	FILE *enc_file = fopen("file.enc", "wb+");
	FILE *dec_file = fopen("file.dec", "wb+");

	// escreve os dados nos respectivos arquivos criados
	fwrite(encrypted_data, data_size_bytes, 1, enc_file); 
	fwrite(decrypted_data, data_size_bytes, 1, dec_file);

	// fecha os arquivos
	fclose(enc_file);
	fclose(dec_file);
};

int main(int argc, char *argv[])
{
	if (argc == 2) {
		enc_dec_file(argv[1]);
	}
	/*
	   printf("Des Tests: %s\n", aes_test() ? "SUCCEEDED" : "FAILED");
	 */
	return(0);
}
