/*********************************************************************
 * Filename:   aes_test.c
 * Author:     Brad Conte (brad AT bradconte.com)
 * Copyright:
 * Disclaimer: This code is presented "as is" without any guarantees.
 * Details:    Performs known-answer tests on the corresponding AES
 implementation. These tests do not encompass the full
 range of available test vectors and are not sufficient
 for FIPS-140 certification. However, if the tests pass
 it is very, very likely that the code is correct and was
 compiled properly. This code also serves as
 example usage of the functions.
 *********************************************************************/

/*************************** HEADER FILES ***************************/
#include <stdio.h>
#include <stdlib.h>
#include <memory.h>
#include <sys/stat.h>
#include "aes.h"
#include <hip/hip_runtime.h>


/*********************** FUNCTION DEFINITIONS ***********************/
void print_hex(BYTE str[], int len)
{
	int idx;

	for(idx = 0; idx < len; idx++)
		printf("%02x", str[idx]);
}

/*int aes_ecb_test()
  {
  WORD key_schedule[60], idx;
  BYTE enc_buf[128];
  BYTE plaintext[2][16] = {
  {0x6b,0xc1,0xbe,0xe2,0x2e,0x40,0x9f,0x96,0xe9,0x3d,0x7e,0x11,0x73,0x93,0x17,0x2a},
  {0xae,0x2d,0x8a,0x57,0x1e,0x03,0xac,0x9c,0x9e,0xb7,0x6f,0xac,0x45,0xaf,0x8e,0x51}
  };
  BYTE ciphertext[2][16] = {
  {0xf3,0xee,0xd1,0xbd,0xb5,0xd2,0xa0,0x3c,0x06,0x4b,0x5a,0x7e,0x3d,0xb1,0x81,0xf8},
  {0x59,0x1c,0xcb,0x10,0xd4,0x10,0xed,0x26,0xdc,0x5b,0xa7,0x4a,0x31,0x36,0x28,0x70}
  };
  BYTE key[1][32] = {
  {0x60,0x3d,0xeb,0x10,0x15,0xca,0x71,0xbe,0x2b,0x73,0xae,0xf0,0x85,0x7d,0x77,0x81,0x1f,0x35,0x2c,0x07,0x3b,0x61,0x08,0xd7,0x2d,0x98,0x10,0xa3,0x09,0x14,0xdf,0xf4}
  };
  int pass = 1;

// Raw ECB mode.
//printf("* ECB mode:\n");
aes_key_setup(key[0], key_schedule, 256);
//printf(  "Key          : ");
//print_hex(key[0], 32);

for(idx = 0; idx < 2; idx++) {
aes_encrypt(plaintext[idx], enc_buf, key_schedule, 256);
//printf("\nPlaintext    : ");
//print_hex(plaintext[idx], 16);
//printf("\n-encrypted to: ");
//print_hex(enc_buf, 16);
pass = pass && !memcmp(enc_buf, ciphertext[idx], 16);

aes_decrypt(ciphertext[idx], enc_buf, key_schedule, 256);
//printf("\nCiphertext   : ");
//print_hex(ciphertext[idx], 16);
//printf("\n-decrypted to: ");
//print_hex(enc_buf, 16);
pass = pass && !memcmp(enc_buf, plaintext[idx], 16);

//printf("\n\n");
}

return(pass);
}*/

int aes_test()
{
	int pass = 1;

	//pass = pass && aes_ecb_test();
	//pass = pass && aes_cbc_test();
	//pass = pass && aes_ctr_test();
	//pass = pass && aes_ccm_test();

	return(pass);
}

void enc_dec_file(char *filename)
{
	/*********************** ABERTURA E LEITURA DO ARQUIVO DE ENTRADA ***********************/
	BYTE *data;
	BYTE *encrypted_data;
	BYTE *decrypted_data;
	//char *filename = "../../sample_files/hubble_1.tif";

	WORD key_schedule[60];

	BYTE key[1][32] = {
		{0x60,0x3d,0xeb,0x10,0x15,0xca,0x71,0xbe,0x2b,0x73,0xae,0xf0,0x85,0x7d,0x77,0x81,0x1f,0x35,0x2c,0x07,0x3b,0x61,0x08,0xd7,0x2d,0x98,0x10,0xa3,0x09,0x14,0xdf,0xf4}
	};

	struct stat st; // stat guarda informações sobre arquivos
	size_t data_size_bytes = 0;

	if (stat(filename, &st) == 0){ // provavelmente verifica se é possivel atribuir o stat st ao arquivo base, e caso sim
		data_size_bytes = sizeof(BYTE) * st.st_size;
		data = (BYTE *) malloc(data_size_bytes); // reserva o tamanho do arquivo base em memória para o ponteiro data.
	};

	FILE *file = fopen(filename, "rb"); // abre o arquivo do tipo nao texto ("rb")

	// copia a imagem para o vetor data
	if(data != NULL && file){ // se o ponteiro data e de arquivo ja foram alocados e abertos, respectivamente, entao prossegue
		int current_byte = 0; // define o byte atual

		// de byte em byte, copia o arquivo para o array data
		while(fread(&data[current_byte], sizeof(BYTE), 1, file) == 1){ 
			current_byte += 1; // atualiza o byte atual
		};
	};

	encrypted_data = (BYTE *) malloc(data_size_bytes); // reserva espaço em memoria para o arquivo criptogrfado
	decrypted_data = (BYTE *) malloc(data_size_bytes); // o mesmo para o arquivo decodificado

	BYTE *d_data; // ponteiro para os dados da imagem base no device
	BYTE *d_encrypted_data; // ponteiro para os dados criptografados no device
	BYTE *d_decrypted_data; // ponteiro para os dados descriptografados no device

	// alocação de memória para os devices
	hipMalloc((void **)&d_data, data_size_bytes);
	hipMalloc((void **)&d_encrypted_data, data_size_bytes);
	hipMalloc((void **)&d_decrypted_data, data_size_bytes);

	hipMemcpy(d_data, data, data_size_bytes, hipMemcpyHostToDevice); // copia o array de dados para o device

	WORD *d_schedule;
	hipMalloc((void **)&d_schedule, sizeof(key_schedule));

	int threadsPerBlock = 256;
	int blocksPerGrid =((data_size_bytes + threadsPerBlock - 1) / threadsPerBlock)/AES_BLOCK_SIZE;

	aes_key_setup(key[0], key_schedule, 256);
	hipMemcpy(d_schedule, key_schedule, sizeof(key_schedule), hipMemcpyHostToDevice);
	aes_encrypt<<<blocksPerGrid, threadsPerBlock>>>(d_data, d_encrypted_data, d_schedule, 256, data_size_bytes); // criptografa o buffer e salva no data_enc
	aes_decrypt<<<blocksPerGrid, threadsPerBlock>>>(d_encrypted_data, d_decrypted_data, d_schedule, 256, data_size_bytes); // descriptografa o data_enc e salva no data_dec

	hipMemcpy(encrypted_data, d_encrypted_data, data_size_bytes, hipMemcpyDeviceToHost); 
	hipMemcpy(decrypted_data, d_decrypted_data, data_size_bytes, hipMemcpyDeviceToHost);

	// cria arquivos para a escrita dos dados cripto e descripto
	FILE *enc_file = fopen("file.enc", "wb+");
	FILE *dec_file = fopen("file.dec", "wb+");

	// escreve os dados nos respectivos arquivos criados
	fwrite(encrypted_data, sizeof(BYTE) * st.st_size, 1, enc_file); 
	fwrite(decrypted_data, sizeof(BYTE) * st.st_size, 1, dec_file);

	// fecha os arquivos
	fclose(enc_file);
	fclose(dec_file);
};

int main(int argc, char *argv[])
{
	if (argc == 2) {
		enc_dec_file(argv[1]);
	}
	/*
	   printf("AES Tests: %s\n", aes_test() ? "SUCCEEDED" : "FAILED");
	 */
	return(0);
}
