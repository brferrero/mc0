/*********************************************************************
 * Filename:   blowfish_test.c
 * Author:     Brad Conte (brad AT bradconte.com)
 * Copyright:
 * Disclaimer: This code is presented "as is" without any guarantees.
 * Details:    Performs known-answer tests on the corresponding Blowfish
 implementation. These tests do not encompass the full
 range of available test vectors, however, if the tests
 pass it is very, very likely that the code is correct
 and was compiled properly. This code also serves as
 example usage of the functions.
 *********************************************************************/

/*************************** HEADER FILES ***************************/
#include <stdio.h>
#include <stdlib.h>
#include <memory.h>
#include <sys/stat.h>
#include "blowfish.h"
#include <hip/hip_runtime.h>

/*********************** FUNCTION DEFINITIONS ***********************/
/*int blowfish_test()
  {
  BYTE key1[8]  = {0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00};
  BYTE key2[8]  = {0xff,0xff,0xff,0xff,0xff,0xff,0xff,0xff};
  BYTE key3[24] = {0xF0,0xE1,0xD2,0xC3,0xB4,0xA5,0x96,0x87,
  0x78,0x69,0x5A,0x4B,0x3C,0x2D,0x1E,0x0F,
  0x00,0x11,0x22,0x33,0x44,0x55,0x66,0x77};
  BYTE p1[BLOWFISH_BLOCK_SIZE] = {0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00};
  BYTE p2[BLOWFISH_BLOCK_SIZE] = {0xff,0xff,0xff,0xff,0xff,0xff,0xff,0xff};
  BYTE p3[BLOWFISH_BLOCK_SIZE] = {0xFE,0xDC,0xBA,0x98,0x76,0x54,0x32,0x10};

  BYTE c1[BLOWFISH_BLOCK_SIZE] = {0x4e,0xf9,0x97,0x45,0x61,0x98,0xdd,0x78};
  BYTE c2[BLOWFISH_BLOCK_SIZE] = {0x51,0x86,0x6f,0xd5,0xb8,0x5e,0xcb,0x8a};
  BYTE c3[BLOWFISH_BLOCK_SIZE] = {0x05,0x04,0x4b,0x62,0xfa,0x52,0xd0,0x80};

  BYTE enc_buf[BLOWFISH_BLOCK_SIZE];
  BLOWFISH_KEY key;
  int pass = 1;

// Test vector 1.
blowfish_key_setup(key1, &key, BLOWFISH_BLOCK_SIZE);
blowfish_encrypt(p1, enc_buf, &key);
pass = pass && !memcmp(c1, enc_buf, BLOWFISH_BLOCK_SIZE);
blowfish_decrypt(c1, enc_buf, &key);
pass = pass && !memcmp(p1, enc_buf, BLOWFISH_BLOCK_SIZE);

// Test vector 2.
blowfish_key_setup(key2, &key, BLOWFISH_BLOCK_SIZE);
blowfish_encrypt(p2, enc_buf, &key);
pass = pass && !memcmp(c2, enc_buf, BLOWFISH_BLOCK_SIZE);
blowfish_decrypt(c2, enc_buf, &key);
pass = pass && !memcmp(p2, enc_buf, BLOWFISH_BLOCK_SIZE);

// Test vector 3.
blowfish_key_setup(key3, &key, 24);
blowfish_encrypt(p3, enc_buf, &key);
pass = pass && !memcmp(c3, enc_buf, BLOWFISH_BLOCK_SIZE);
blowfish_decrypt(c3, enc_buf, &key);
pass = pass && !memcmp(p3, enc_buf, BLOWFISH_BLOCK_SIZE);

return(pass);
}*/

void enc_dec_file(char *filename)
{
	/*********************** ABERTURA E LEITURA DO ARQUIVO DE ENTRADA ***********************/
	BYTE *data;
	BYTE *encrypted_data;
	BYTE *decrypted_data;
	//char *filename = "../../sample_files/king_james_bible.txt";

	BYTE key1[8]  = {0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00};
	/*BYTE key2[8]  = {0xff,0xff,0xff,0xff,0xff,0xff,0xff,0xff};
	  BYTE key3[24] = {0xF0,0xE1,0xD2,0xC3,0xB4,0xA5,0x96,0x87,
	  0x78,0x69,0x5A,0x4B,0x3C,0x2D,0x1E,0x0F,
	  0x00,0x11,0x22,0x33,0x44,0x55,0x66,0x77};*/

	BLOWFISH_KEY key;

	struct stat st;
	size_t data_size_bytes = 0;

	if (stat(filename, &st) == 0) { // provavelmente verifica se é possivel atribuir o stat st ao arquivo base, e caso sim
		data_size_bytes = sizeof(BYTE) * st.st_size;
		data = (BYTE *) malloc(data_size_bytes); // reserva o tamanho do arquivo base em memória para o ponteiro data.
	};

	FILE *file = fopen(filename, "rb");

	if(data != NULL && file){
		int current_byte = 0;

		while(fread(&data[current_byte], sizeof(BYTE), 1, file) == 1){
			current_byte += 1;
		};
	};

	encrypted_data = (BYTE *) malloc(data_size_bytes); // reserva espaço em memoria para o arquivo criptogrfado
	decrypted_data = (BYTE *) malloc(data_size_bytes); // o mesmo para o arquivo decodificado

	BYTE *d_data; // ponteiro para os dados da imagem base no device
	BYTE *d_encrypted_data; // ponteiro para os dados criptografados no device
	BYTE *d_decrypted_data; // ponteiro para os dados descriptografados no device

	// alocação de memória para os devices
	hipMalloc((void **)&d_data, data_size_bytes);
	hipMalloc((void **)&d_encrypted_data, data_size_bytes);
	hipMalloc((void **)&d_decrypted_data, data_size_bytes);

	hipMemcpy(d_data, data, data_size_bytes, hipMemcpyHostToDevice); // copia o array de dados para o device

	BLOWFISH_KEY *d_key;
	hipMalloc((void **)&d_key, sizeof(BLOWFISH_KEY));
	blowfish_key_setup(key1, &key, BLOWFISH_BLOCK_SIZE);
	hipMemcpy(d_key, &key, sizeof(BLOWFISH_KEY), hipMemcpyHostToDevice);

	int threadsPerBlock = 256;
	int blocksPerGrid = ((data_size_bytes + threadsPerBlock - 1) / threadsPerBlock)/BLOWFISH_BLOCK_SIZE;

	blowfish_encrypt<<<blocksPerGrid, threadsPerBlock>>>(d_data, d_encrypted_data, d_key, data_size_bytes);
	blowfish_decrypt<<<blocksPerGrid, threadsPerBlock>>>(d_encrypted_data, d_decrypted_data, d_key, data_size_bytes);

	hipMemcpy(encrypted_data, d_encrypted_data, data_size_bytes, hipMemcpyDeviceToHost); 
	hipMemcpy(decrypted_data, d_decrypted_data, data_size_bytes, hipMemcpyDeviceToHost);

	FILE *enc_file = fopen("file.enc", "wb+");
	FILE *dec_file = fopen("file.dec", "wb+");

	fwrite(encrypted_data, sizeof(BYTE) * st.st_size, 1, enc_file);
	fwrite(decrypted_data, sizeof(BYTE) * st.st_size, 1, dec_file);

	fclose(enc_file);
	fclose(dec_file);
};

int main(int argc, char *argv[])
{
	if (argc == 2) {
		enc_dec_file(argv[1]);
	}
	/*
	   printf("Blowfish Tests: %s\n", aes_test() ? "SUCCEEDED" : "FAILED");
	 */
	return(0);
}
